#include "graph.h"
#include "ATD.h"

int main(int argc, char* argv[]) { 
    GPUGraph *g;
    hipMallocManaged(&g, sizeof(GPUGraph));
    g->readBinaryFile(argv[1]);
    hipDeviceSynchronize();
    floyd_warshall_kernel<<<BLK_NUMS, BLK_DIM>>>(g);
    hipDeviceSynchronize();
    return 0;
}