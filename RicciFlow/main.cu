#include "graph.h"
#include "ATD.h"

int main(int argc, char* argv[]) { 
    GPUGraph *g;
    hipMallocManaged(&g, sizeof(GPUGraph));
    g->readBinaryFile(argv[1]);
    floyd_warshall_kernel<<<BLK_NUMS, BLK_DIM>>>(g);
    printf("\n");
    return 0;
}