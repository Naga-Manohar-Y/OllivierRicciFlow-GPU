#include "hip/hip_runtime.h"
#include "common.h"
#include "graph.h"
#include "ATD.h"
__global__ d_update_weights(GPUGraph* g, float* edge_RC){
    float sumw = 0;
    for(ui e=GLTHID; e<M; e+=N_THREADS){
        g->d_weights[e]-=EPSILON*edge_RC[e]*g->d_weights[e];
        sumw+=g->d_weights[e];
    } 
    warp_sum(sumw);
    if (LANEID==0)
    atomicAdd(gsum, sumw);
}
__global__ d_normalize_weights(GPUGraph* g, float* edge_RC){
    for(ui e=GLTHID; e<M; e+=N_THREADS){
        g->d_weights[e]*=(m/gsum[0]);
    }
}

class RicciFlow{
    GPUGraph* g;
    float* gsum;
    float* apsp;
    float* edge_RC;
    float* node_RC;
    string method="ATD";
public:
    RicciFlow(GPUGraph *_g){
        g=_g;
        hipMallocManaged(&apsp, sizeof(float)*N*N);
        hipMallocManaged(&edge_RC, sizeof(float)*M);
        hipMallocManaged(&node_RC, sizeof(float)*N);
        hipMallocManaged(&gsum, sizeof(float));
    }
    void compute_edge_RC(){
        if (method=="ATD"){
            compute_RC_ATD(g, apsp, edge_RC);
        }
        else if(method=="OTD"){
            compute_RC_OTD(g, apsp, edge_RC);
        }
        else if(method=="Sinkhorn"){
            compute_RC_Sinkhorn(g, apsp, edge_RC);
        }
        else{
            cout<<"Not a valid method"<<endl;
            return;
        }
    }
    void update_weights(){
        *gsum = 0;
        d_update_weights<<<BLK_NUMS, BLK_DIM>>>(g, edge_RC);
        d_normalize_weights<<<BLK_NUMS, BLK_DIM>>>(g);
        hipDeviceSynchronize();
    }
    void ricci_flow(){

        for(ui i=0;i<N_ITER; i++){
            compute_edge_RC(g, apsp, edge_RC);
            // compute_node_RC(g, edge_RC, node_RC);
            update_weights(g, edge_RC); // step 3, 4 of algo
            // todo check the condition cruvatuer values do not change a lot
        }
    }
};
    
int main(int argc, char* argv[]) { 
    GPUGraph *g;
    hipMallocManaged(&g, sizeof(GPUGraph));
    g->readBinaryFile(argv[1]);
    g->copyToGPU();
    RicciFlow rf(g);
    rf.ricci_flow();
    return 0;
}