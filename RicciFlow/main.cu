#include "graph.h"
#include "ATD.h"

int main(int argc, char* argv[]) { 
    GPUGraph *g;
    hipMallocManaged(&g, sizeof(GPUGraph));
    g->readBinaryFile(argv[1]);
    for(ui i=0;i<g->n;i++){
        cout<<g->offset[i]<<" ";
    }
    cout<<endl;
    floyd_warshall_kernel<<<BLK_NUMS, BLK_DIM>>>(g);
    hipDeviceSynchronize();

    // chkerr(hipMemcpy(g->offset, g->d_offset, g->n * sizeof(ui), hipMemcpyDeviceToHost));
    for(ui i=0;i<g->n;i++){
        cout<<g->offset[i]<<" ";
    }
    cout<<endl;
    return 0;
}