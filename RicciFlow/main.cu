#include "graph.h"
#include "ATD.h"

int main(int argc, char* argv[]) { 
    GPUGraph *g;
    hipMallocManaged(&g, sizeof(GPUGraph));
    g->readBinaryFile(argv[1]);
    floyd_warshall_kerne<<<BLK_NUMS, BLK_DIM>>>(g);
    return 0;
}