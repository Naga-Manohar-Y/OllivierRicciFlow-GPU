#include "hip/hip_runtime.h"
#include "common.h"
#include "graph.h"
#include "ATD.h"
__device__ float* gsum;

__global__ void d_update_weights(GPUGraph* g, float* edge_RC){
    float sumw = 0;
    for(ui e=GTHID; e<M; e+=N_THREADS){
        g->d_weights[e]-=EPSILON*edge_RC[e]*g->d_weights[e];
        sumw+=g->d_weights[e];
    } 
    warp_sum(sumw);
    if (LANEID==0)
    atomicAdd(gsum, sumw);
}
__global__ void d_normalize_weights(GPUGraph* g){
    for(ui e=GTHID; e<M; e+=N_THREADS){
        g->d_weights[e]*=(M/gsum[0]);
    }
}

class RicciFlow{
    GPUGraph* g;
    float* apsp;
    float* edge_RC;
    float* node_RC;
    string method="ATD";
public:
    RicciFlow(GPUGraph *_g){
        g=_g;
        hipMallocManaged(&apsp, sizeof(float)*N*N);
        hipMalloc(&edge_RC, sizeof(float)*M);
        hipMalloc(&node_RC, sizeof(float)*N);
        hipMallocManaged(&gsum, sizeof(float));
    }
    void compute_edge_RC(){
        if (method=="ATD"){
            compute_RC_ATD(g, apsp, edge_RC);
        }
        // else if(method=="OTD"){
        //     compute_RC_OTD(g, apsp, edge_RC);
        // }
        // else if(method=="Sinkhorn"){
        //     compute_RC_Sinkhorn(g, apsp, edge_RC);
        // }
        else{
            cout<<"Not a valid method"<<endl;
            return;
        }
    }
    void update_weights(){
        hipMemset(gsum, sizeof(float), 0);
        d_update_weights<<<BLK_NUMS, BLK_DIM>>>(g, edge_RC);
        d_normalize_weights<<<BLK_NUMS, BLK_DIM>>>(g);
        hipDeviceSynchronize();
    }
    void ricci_flow(){

        for(ui i=0;i<N_ITER; i++){
            compute_edge_RC();
            // compute_node_RC(g, edge_RC, node_RC);
            update_weights(); // step 3, 4 of algo
            // todo check the condition cruvatuer values do not change a lot
        }
    }
};
    
int main(int argc, char* argv[]) { 
    GPUGraph *g;
    hipMallocManaged(&g, sizeof(GPUGraph));
    g->readBinaryFile(argv[1]);
    g->copyToGPU();
    RicciFlow rf(g);
    rf.ricci_flow();
    return 0;
}