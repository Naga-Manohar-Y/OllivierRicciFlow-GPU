#include "hip/hip_runtime.h"
#include "common.h"
#include "graph.h"
#include "ATD.h"
__managed__ float gsum;

__global__ void d_update_weights(GPUGraph* g, float* edge_RC){
    float sumw = 0;
    for(ui e=GTHID; e<M; e+=N_THREADS){
        g->d_weights[e]-=EPSILON*edge_RC[e]*g->d_weights[e];
        sumw+=g->d_weights[e];
    } 
    warp_sum(sumw);
    if (LANEID==0)
    atomicAdd(&gsum, sumw);
}
__global__ void d_normalize_weights(GPUGraph* g){
    for(ui e=GTHID; e<M; e+=N_THREADS){
        g->d_weights[e]*=(M/gsum);
    }
}

class RicciFlow{
    GPUGraph* g;
    float* apsp;
    float* edge_RC;
    float* node_RC;
    string method="ATD";
public:
    RicciFlow(GPUGraph *_g){
        g=_g;
        hipMalloc(&apsp, sizeof(float)*N*N);
        hipMalloc(&edge_RC, sizeof(float)*M);
        hipMalloc(&node_RC, sizeof(float)*N);
        // hipMallocManaged(&gsum, sizeof(float));
    }
    void compute_edge_RC(){
        if (method=="ATD"){
            compute_RC_ATD(g, apsp, edge_RC);
        }
        // else if(method=="OTD"){
        //     compute_RC_OTD(g, apsp, edge_RC);
        // }
        // else if(method=="Sinkhorn"){
        //     compute_RC_Sinkhorn(g, apsp, edge_RC);
        // }
        else{
            cout<<"Not a valid method"<<endl;
            return;
        }
    }
    void update_weights(){
        gsum = 0;
        // hipMemset(gsum, sizeof(float), 0);
        d_update_weights<<<BLK_NUMS, BLK_DIM>>>(g, edge_RC);
        d_normalize_weights<<<BLK_NUMS, BLK_DIM>>>(g);
        hipDeviceSynchronize();
    }
    void ricci_flow(){

        for(ui i=0;i<N_ITER; i++){
            // compute_edge_RC();
            // compute_node_RC(g, edge_RC, node_RC);
            // update_weights(); // step 3, 4 of algo
            // todo check the condition curvature values do not change a lot
        }
        hipDeviceSynchronize();
        float *weights=new float[M];
        chkerr(hipMemcpy(weights, g->d_weights, M * sizeof(float), hipMemcpyDeviceToHost));
        // chkerr(hipMemcpy(weights, g->d_degree, 10 * sizeof(ui), hipMemcpyDeviceToHost));

        for(ui i=0;i<M;i++){
            cout<<weights[i]<<" ";
        }

    }
};
    
int main(int argc, char* argv[]) { 
    GPUGraph *g;
    hipMallocManaged(&g, sizeof(GPUGraph));
    g->readBinaryFile(argv[1]);
    g->copyToGPU();
    RicciFlow rf(g);
    rf.ricci_flow();
    return 0;
}